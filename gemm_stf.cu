#include <cuda/experimental/stf.cuh>

using namespace cuda::experimental::stf;

int main(int argc, char **argv)
{
    size_t N = 640;
    size_t NBLOCKS = 128;
    size_t NITER = 20;

    if (argc > 1) {
       N = atol(argv[1]);
       fprintf(stderr, "N = %zu\n", N);
    }

    if (argc > 2) {
       NBLOCKS = atol(argv[2]);
       fprintf(stderr, "NBLOCKS = %zu\n", NBLOCKS);
    }

    if (argc > 3) {
       NITER = atol(argv[3]);
       fprintf(stderr, "NITER = %zu\n", NITER);
    }

    hipblasHandle_t handle;
    cuda_safe_call(hipblasCreate(&handle));

    context ctx;
    if (argc > 4) {
        if (atoi(argv[4])) {
           fprintf(stderr, "Using CUDA graphs.\n");
           ctx = graph_ctx();
        }
    }

    std::vector<logical_data<slice<double, 2>>> vA;
    std::vector<logical_data<slice<double, 2>>> vB;
    std::vector<logical_data<slice<double, 2>>> vC;

    for (size_t k = 0; k < NBLOCKS; k++)
    {
        vA.push_back(ctx.logical_data(shape_of<slice<double, 2>>(N, N)));
        vB.push_back(ctx.logical_data(shape_of<slice<double, 2>>(N, N)));
        vC.push_back(ctx.logical_data(shape_of<slice<double, 2>>(N, N)));
    }

    for (size_t k = 0; k < NBLOCKS; k++)
    {
        ctx.parallel_for(vA[k].shape(), vA[k].write(), vB[k].write(), vC[k].write())->*[N]__device__(size_t i, size_t j, auto a, auto b, auto c) {
            a(i, j) = (1.0*(i+j))/N;
            b(i, j) = (1.0*(i-j))/N;
            c(i, j) = 1.0;
        };
    }

    hipEvent_t start, stop;
    cuda_safe_call(hipEventCreate(&start));
    cuda_safe_call(hipEventCreate(&stop));
    cuda_safe_call(hipEventRecord(start, ctx.task_fence()));

    for (size_t iter = 0; iter < NITER; iter++)
    {
        fprintf(stderr, "Iteration %zu\n", iter);

        for (size_t k = 0; k < NBLOCKS; k++) {
            ctx.task(vA[k].read(), vB[k].read(), vC[k].rw())->*[&](hipStream_t stream, auto a, auto b, auto c) {
                    const double alpha = 1.0;
                    const double beta = 1.0;
                    cuda_safe_call(hipblasSetStream(handle, stream));
                    cuda_safe_call(hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N,
                        &alpha, a.data_handle(), N, b.data_handle(), N,
                        &beta,  c.data_handle(), N));
            };
        }

        // As a side effect, it will generate a new CUDA graph
        ctx.task_fence();
    }

    cuda_safe_call(hipEventRecord(stop, ctx.task_fence()));

    ctx.finalize();

    float elapsed;
    cuda_safe_call(hipEventElapsedTime(&elapsed, start, stop));
    fprintf(stderr, "Elapsed : %f ms per iteration.\n", elapsed/NITER);
}
